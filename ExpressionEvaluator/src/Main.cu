#include "hip/hip_runtime.h"
/*
 ============================================================================
 Name        : Main.cu
 Author      : Piotr Luboń
 Version     :
 Copyright   :
 Description :
 ============================================================================
 */
#include <iostream>
#include <numeric>
#include <stdlib.h>
#include <stdio.h>
#include <fstream>
#include <vector>
#include <string>
#include <iostream>
#include <cstdlib>
#include <iomanip>
#include <unistd.h>
#include "ProblemInstance.h"
#include "Evaluator.cuh"
#include <cfloat>
#include <sys/time.h>
#include <ctime>

#define CUDA_CHECK_RETURN(value) CheckCudaErrorAux(__FILE__,__LINE__, #value, value)
#define MAX_BLOCK_SIZE 512
typedef long long int64; typedef unsigned long long uint64;


int main(int argc, char * argv[])
{
	hipEvent_t start, stop, startCopy, endCopy, startSingle, endSingle;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventCreate(&startCopy);
	hipEventCreate(&endCopy);
	hipEventCreate(&startSingle);
	hipEventCreate(&endSingle);
	char cwd[1024];
	if (getcwd(cwd, sizeof(cwd)) != NULL)
	       fprintf(stdout, "Current working dir: %s\n", cwd);
	string filename="testfile";
	ProblemInstance problem(filename);
	float finalResult[problem.length];
	char* devExpression = NULL;
	float* devVectors = NULL;
	float* devResult = NULL;
	hipEventRecord(startCopy, 0);
	problem.CopyToDevice(devExpression, devVectors, devResult);
	hipEventRecord(endCopy, 0);
	hipEventSynchronize(endCopy);
	float elapsedCopy;
	hipEventElapsedTime(&elapsedCopy, startCopy, endCopy);
	int explen = problem.ExpLength;
	int width = (explen+1)/4;
	int threads = MAX_BLOCK_SIZE-(MAX_BLOCK_SIZE%width);
	int blocks = ((problem.length * width) / threads);
	if(problem.length*width%threads!=0)
		blocks++;
	int aligned = problem.GetNumOfVariables();
	if(aligned%4!=0)
			aligned = problem.GetNumOfVariables() + 4 - (problem.GetNumOfVariables()%4);
	size_t sharedPerBlock = (threads + (aligned * (MAX_BLOCK_SIZE/width)))  * sizeof(float)+ ((explen+1) * sizeof(char));
	cout<<"Problem length: "<<problem.length<<endl;
	cout<<"Array tree length: "<<explen<<endl;
	cout<<"Threads per subproblem: "<<width<<endl;
	cout<<"Threads per block: "<<threads<<endl;
	cout<<"Blocks: "<<blocks<<endl;
	cout<<"Bytes of shared memory per block: "<<sharedPerBlock<<endl;
	hipEventRecord(start, 0);
	Evaluate<<<blocks, MAX_BLOCK_SIZE, sharedPerBlock>>>(devVectors, devResult, devExpression, problem.length, problem.GetNumOfVariables(), explen);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	float elapsedTime;
	hipEventElapsedTime(&elapsedTime, start, stop);
	hipEventDestroy(start);
	hipEventDestroy(stop);
	hipMemcpy(finalResult, devResult, problem.length*sizeof(float), hipMemcpyDeviceToHost);
	struct timeval t5, t6;
	gettimeofday(&t5, NULL);
	vector<float> cpuResults = problem.EvaluateCpu();
	gettimeofday(&t6, NULL);
	uint64 diff = ((t6.tv_sec - t5.tv_sec) * 1000) +(t6.tv_usec/1000 - t5.tv_usec/1000);
	bool ok = true;
	for(int i = 0; i < problem.length; i++)
	{
		if(!(fabs(cpuResults[i] - finalResult[i]) < FLT_EPSILON))
		{
			cout<<i<<" "<<cpuResults[i]<<" : "<<finalResult[i]<<endl;
			ok = false;
			break;
		}
	}
	if(ok)
		cout<<"Results ok"<<endl;
	else
		cout<<"Results not ok"<<endl;
	memset(finalResult, 0, problem.length * sizeof(float));
	hipFree(devExpression);
	hipFree(devVectors);
	hipFree(devResult);
	cout<<"Copying time: "<<elapsedCopy<<endl;
	cout<<"Calculation time: "<<elapsedTime<<endl;
	cout<<"Cpu calculation time:"<<diff<<endl;
	hipFree(devExpression);
	hipFree(devVectors);
	hipFree(devResult);
}



